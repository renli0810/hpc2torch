#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

constexpr long long BLOCKdim = 128;
// constexpr long long GRIDdim_x =  ;
// constexpr long long GRIDdim_y = 1;

template <typename T, typename Tind>
__global__ void gatherkernel(T const *input, Tind const *indices, T *output, long long stride, long long allSize)
{
    long long overcount = (stride + BLOCKdim - 1) / BLOCKdim;
    long long outputindex = blockIdx.x * gridDim.y * stride + blockIdx.y * stride + threadIdx.x;
    long long index = indices[blockIdx.y];
    long long inputindex = blockIdx.x * gridDim.y * stride + index * stride + threadIdx.x;
    if (threadIdx.x >= stride)
        return;
    for (int i = 0; i < overcount; i++)
    {
        if (outputindex >= allSize)
        {
            return;
        }
        output[outputindex] = input[inputindex];
        // printf("%d %d %d %d stride:%d\n",gridDim.x,gridDim.y,blockDim.x, blockDim.y, stride);
        // printf("outputindex=%d inputindex=%d input[inputindex]=%lf index=%d allSize=%d blockx=%d blocky=%d threadx=%d\n", outputindex, inputindex, input[inputindex], index, allSize, blockIdx.x, blockIdx.y, threadIdx.x);
        inputindex += BLOCKdim;
        outputindex += BLOCKdim;
    }
}

template <typename T, typename Tind>
void gatherLaunch(void const *input, void const *indices, void *output, long long stride, long long axisSize, long long inputSize, long long indexSize)
{
    long long griddim_x = inputSize / axisSize / stride;
    long long griddim_y = indexSize;
    long long allSize = indexSize * inputSize / axisSize;
    // printf("x:%d y:%d\n", griddim_x, griddim_y);
    dim3 GRIDdim(griddim_x, griddim_y);
    gatherkernel<T, Tind>
        <<<GRIDdim, BLOCKdim>>>((T *)input, (Tind *)indices, (T *)output, stride, allSize);
}
extern "C" void gather_nv_f32(void const *input, void const *indices, void *output, long long stride, long long axisSize, long long inputSize, long long indexSize)
{
    gatherLaunch<float, uint64_t>(input, indices, output, stride, axisSize, inputSize, indexSize);
}
extern "C" void gather_nv_f16(void const *input, void const *indices, void *output, long long stride, long long axisSize, long long inputSize, long long indexSize)
{
    gatherLaunch<half, uint64_t>(input, indices, output, stride, axisSize, inputSize, indexSize);
}